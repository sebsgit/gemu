#include <iostream>
#include <string>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void kernel(int * out){
	*out = 5;
}

int main(){
	int * result;
	hipMalloc(&result, sizeof(*result));
	kernel<<< dim3(1,1,1), dim3(1,1,1) >>> (result);
	int hostResult=0;
	hipMemcpy(&hostResult, result, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(result);
	std::cout << hostResult << '\n';
	return 0;
}
